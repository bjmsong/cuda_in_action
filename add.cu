
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__
void add(int *a, int *b, int *c)
{
    // 这边的*是解引用操作符
    *c = *a + *b;
}


int main(void){
    int a = 1, b = 2, c;
    int *d_a, *d_b, *d_c;  // device copies of a,b,c

    // Allocate space for device copies of a,b,c
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << a << " " << b << " " << c << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}